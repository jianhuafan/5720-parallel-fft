#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void ComplexMul(hipfftComplex *a, hipfftComplex *b, int size) {
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads) {
        hipfftComplex c;
        c.x = a[i].x * b[i].x - a[i].y*b[i].y;
        c.y = a[i].x * b[i].y + a[i].y*b[i].x;
        c.x /= size;
        c.y /= size;
        a[i] = c;
    }
}

// Pad data
int PadData(const hipfftComplex *signal, hipfftComplex **padded_signal, int signal_size,
            const hipfftComplex *filter_kernel, hipfftComplex **padded_filter_kernel,
            int filter_kernel_size) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;
  int new_size = signal_size + maxRadius;

  // Pad signal
  hipfftComplex *new_data =(hipfftComplex *)(malloc(sizeof(hipfftComplex) * new_size));
  memcpy(new_data + 0, signal, signal_size * sizeof(hipfftComplex));
  memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(hipfftComplex));
  *padded_signal = new_data;

  // Pad filter
  new_data = (hipfftComplex *)(malloc(sizeof(hipfftComplex) * new_size));
  memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(hipfftComplex));
  memset(new_data + maxRadius, 0,
         (new_size - filter_kernel_size) * sizeof(hipfftComplex));
  memcpy(new_data + new_size - minRadius, filter_kernel,
         minRadius * sizeof(hipfftComplex));
  *padded_filter_kernel = new_data;

  return new_size;
}

void feed_gaussian_kernel(hipfftComplex *filter_kernel, int filter_kernel_size) {
    for (int i = 0; i < filter_kernel_size; i++) {
        filter_kernel[i].x = 0.0;
        filter_kernel[i].y = 0.0;
    }
    filter_kernel[0].x = 1.0f / 16;
    filter_kernel[1].x = 2.0f / 16;
    filter_kernel[2].x = 1.0f / 16;
    filter_kernel[3].x = 2.0f / 16;
    filter_kernel[4].x = 4.0f / 16;
    filter_kernel[5].x = 2.0f / 16;
    filter_kernel[6].x = 1.0f / 16;
    filter_kernel[7].x = 2.0f / 16;
    filter_kernel[8].x = 1.0f / 16;
}

void feed_identity_kernel(hipfftComplex *filter_kernel, int filter_kernel_size) {
    for (int i = 0; i < filter_kernel_size; i++) {
        filter_kernel[i].x = 0.0;
        filter_kernel[i].y = 0.0;
        if (i == filter_kernel_size / 2) {
            filter_kernel[i].x = 1.0;
        }
    }
}

void feed_edge_detection_kernel(hipfftComplex *filter_kernel, int filter_kernel_size) {
    for (int i = 0; i < filter_kernel_size; i++) {
        filter_kernel[i].x = 0.0;
        filter_kernel[i].y = 0.0;
    }
    filter_kernel[0].x = -1.0f;
    filter_kernel[1].x = -1.0f;
    filter_kernel[2].x = -1.0f;
    filter_kernel[3].x = -1.0f;
    filter_kernel[4].x = 8.0f;
    filter_kernel[5].x = -1.0f;
    filter_kernel[6].x = -1.0f;
    filter_kernel[7].x = -1.0f;
    filter_kernel[8].x = -1.0f;
}

void feed_box_blur_kernel(hipfftComplex *filter_kernel, int filter_kernel_size) {
    for (int i = 0; i < filter_kernel_size; i++) {
        filter_kernel[i].x = 1.0 / filter_kernel_size;
        filter_kernel[i].y = 0.0;
    }
}

void feed_sharpen_kernel(hipfftComplex *filter_kernel, int filter_kernel_size) {
    for (int i = 0; i < filter_kernel_size; i++) {
        filter_kernel[i].x = 0.0;
        filter_kernel[i].y = 0.0;
    }
    filter_kernel[1].x = -1.0f;
    filter_kernel[3].x = -1.0f;
    filter_kernel[4].x = 5.0f;
    filter_kernel[5].x = -1.0f;
    filter_kernel[7].x = -1.0f;
}

int main(int argc, char **argv) {

    // load image
    int width, height, bpp;
    uint8_t* rgb_image = stbi_load("input/256.png", &width, &height, &bpp, STBI_grey);

    int filter_kernel_size = 15 * 15;

    float elapsedTime = 0;
    hipfftHandle plan;
    int signal_size = width * height * sizeof(hipfftComplex);

    hipfftComplex *signal = (hipfftComplex*)malloc(signal_size);
    hipfftComplex *filter_kernel = (hipfftComplex*)malloc(filter_kernel_size * sizeof(hipfftComplex));
    hipfftComplex *dev_signal;
    hipfftComplex *dev_filter_kernel;
    hipEvent_t start, stop;

    // feed input
    srand(time(NULL));
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            uint8_t* pixel = rgb_image + (i * width + j);
            signal[i * width + j].x = (float)pixel[0];
            signal[i * width + j].y = 0.0;
        }
    }

    // feed kernel
    feed_identity_kernel(filter_kernel, filter_kernel_size);

    // pad image and filter kernel
    hipfftComplex *padded_signal;
    hipfftComplex *padded_filter_kernel;
    int new_size = PadData(signal, &padded_signal, width * height, filter_kernel,
              &padded_filter_kernel, filter_kernel_size);
    
    int mem_size = sizeof(hipfftComplex) * new_size;

    // allocate gpu memory
    hipMalloc((void**)&dev_signal, mem_size);
    hipMalloc((void**)&dev_filter_kernel, mem_size);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy padded input to device
    hipMemcpy(dev_signal, padded_signal, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_filter_kernel, padded_filter_kernel, mem_size, hipMemcpyHostToDevice);
    
    // create cufft plan
    hipfftPlan2d(&plan, height, width, HIPFFT_C2C);

    // perform 2dfft
    hipfftExecC2C(plan, dev_signal, dev_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, dev_filter_kernel, dev_filter_kernel, HIPFFT_FORWARD);

    // perform multiplication
    ComplexMul <<<32, 256>>>(dev_signal, dev_filter_kernel, new_size);

    // perform inverse 2dfft
    hipfftExecC2C(plan, dev_signal, dev_signal, HIPFFT_BACKWARD);

    // copy back results
    hipfftComplex *convolved_signal = padded_signal;
    hipMemcpy(convolved_signal, dev_signal, mem_size, hipMemcpyDeviceToHost);

    // get calculation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // show results
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            printf("DATA: %3.1f %3.1f \n", convolved_signal[i * 4 + j].x, convolved_signal[i * 4 + j].y);
        }
    }
    printf("CUFFT calculation completed: %5.3f ms\n", elapsedTime);

    // write filtered image
    uint8_t* output_rgb_image;
    output_rgb_image = (uint8_t*)malloc(width*height);
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            output_rgb_image[i * width + j] = (uint8_t)convolved_signal[i * width + j].x;
            if (i < 4 && j < 4) {
                printf("%hhu\n", output_rgb_image[i * width + j]);
            }
        }
    }

    stbi_write_png("output/cuda/filtered_identity_256.png", width, height, 1, output_rgb_image, width);

    // free memory
    hipfftDestroy(plan);
    hipFree(dev_signal);
    hipFree(dev_filter_kernel);
    free(signal);
    free(filter_kernel);
    stbi_image_free(rgb_image);

    return 0;
}
