#include "hip/hip_runtime.h"
// compile command: /usr/local/cuda-10.0/bin/nvcc -arch=compute_35 cuda_fft.cu -lcublas

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef float2 Complex;

#define DIM 65536

int main(int argc, char **argv) {
    float elapsedTime = 0;
    hipfftHandle plan;
    hipfftComplex *host_data = (hipfftComplex*)malloc(DIM * sizeof(hipfftComplex));
    hipfftComplex *dev_data;
    hipEvent_t start, stop;

    // feed input
    srand(time(NULL));
    for (int i = 0; i < DIM; i++) {
        host_data[i].x = rand() / (float) RAND_MAX;
        host_data[i].y = 0.0;
    }

    // allocate gpu memory
    hipMalloc((void**)&dev_data, sizeof(hipfftComplex) * DIM);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy input to device
    hipMemcpy(dev_data, host_data, DIM * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    
    // create cufft plan
    hipfftPlan1d(&plan, DIM, CUFFT_C22, 1);

    // perform computation
    hipfftExecC2C(plan, dev_data, dev_data, HIPFFT_FORWARD);

    // copy back results
    hipMemcpy(host_data, dev_data, sizeof(hipfftComplex) * DIM, hipMemcpyDeviceToHost);

    // get calculation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // show results
    for (int i = 0; i < 16; i++) {
        printf("DATA: %3.1f %3.1f \ n", host_data[i].x, host_data[i].y);
    }

    // free memory
    hipfftDestroy(plan);
    hipFree(dev_data);
    free(host_data);
    
    return 0;
}