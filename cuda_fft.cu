#include "hip/hip_runtime.h"
// compile command: /usr/local/cuda-10.0/bin/nvcc -arch=compute_35 cuda_fft.cu -lcublas -lcufft

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef float2 Complex;

#define N 4

int main(int argc, char **argv) {
    float elapsedTime = 0;
    hipfftHandle plan;
    hipfftComplex *host_data = (hipfftComplex*)malloc(N*N * sizeof(hipfftComplex));
    hipfftComplex *dev_data;
    hipEvent_t start, stop;

    // feed input
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; k++) {
            host_data[i * N + j].x = rand() / (float) RAND_MAX;
            host_data[i * N + j].y = 0.0;
        }
    }

    // allocate gpu memory
    hipMalloc((void**)&dev_data, sizeof(hipfftComplex) * N*N);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy input to device
    hipMemcpy(dev_data, host_data, N*N * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    
    // create cufft plan
    cufftPlan2D(&plan, N, N, HIPFFT_C2C, 1);

    // perform computation
    hipfftExecC2C(plan, dev_data, dev_data, HIPFFT_FORWARD);

    // copy back results
    hipMemcpy(host_data, dev_data, sizeof(hipfftComplex) * N*N, hipMemcpyDeviceToHost);

    // get calculation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // show results
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("DATA: %3.1f %3.1f \n", host_data[i].x, host_data[i].y);
        }
    }
    printf("CUFFT calculation completed: %5.3f ms\n", elapsedTime);

    // free memory
    hipfftDestroy(plan);
    hipFree(dev_data);
    free(host_data);

    return 0;
}