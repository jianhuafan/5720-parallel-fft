#include "hip/hip_runtime.h"
// compile command: /usr/local/cuda-10.0/bin/nvcc -arch=compute_35 cuda_fft.cu -lcublas -lcufft

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void ComplexMul(hipfftComplex *a, hipfftComplex *b, int size) {
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads) {
        hipfftComplex c;
        c.x = a[i].x * b[i].x - a[i].y*b[i].y;
        c.y = a[i].x * b[i].y + a[i].y*b[i].x;
        c.x /= (1.0f / size);
        c.y /= (1.0f / size);
        a[i] = c;
    }
}

int main(int argc, char **argv) {

    // load image
    int width, height, bpp;
    uint8_t* rgb_image = stbi_load("image/dog.jpg", &width, &height, &bpp, STBI_grey);

    float elapsedTime = 0;
    hipfftHandle plan;
    int mem_size = width * height * sizeof(hipfftComplex);

    hipfftComplex *signal = (hipfftComplex*)malloc(mem_size);
    hipfftComplex *filter_kernel = (hipfftComplex*)malloc(mem_size);
    hipfftComplex *dev_signal;
    hipfftComplex *dev_filter_kernel;
    hipEvent_t start, stop;

    // feed input
    srand(time(NULL));
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            uint8_t* pixel = rgb_image + (i * width + j);
            signal[i * width + j].x = (float)pixel[0];
            signal[i * width + j].y = 0.0;
        }
    }

    // feed kernel
    for (int i = 0; i < height * width; i++) {
        filter_kernel[i].x = rand() / (float) RAND_MAX;
        filter_kernel[i].y = 0.0;
    }

    // allocate gpu memory
    hipMalloc((void**)&dev_signal, mem_size);
    hipMalloc((void**)&dev_filter_kernel, mem_size);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy input to device
    hipMemcpy(dev_signal, signal, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_filter_kernel, filter_kernel, mem_size, hipMemcpyHostToDevice);
    
    // create cufft plan
    hipfftPlan2d(&plan, height, width, HIPFFT_C2C);

    // perform 2dfft
    hipfftExecC2C(plan, dev_signal, dev_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, dev_filter_kernel, dev_filter_kernel, HIPFFT_FORWARD);

    // perform multiplication
    ComplexMul <<<32, 256>>>(dev_signal, dev_filter_kernel, width * height);

    // perform inverse 2dfft
    hipfftExecC2C(plan, dev_signal, dev_signal, HIPFFT_BACKWARD);

    // copy back results
    hipMemcpy(signal, dev_signal, mem_size, hipMemcpyDeviceToHost);

    // get calculation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // show results
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            printf("DATA: %3.1f %3.1f \n", signal[i * 4 + j].x, signal[i * 4 + j].y);
        }
    }
    printf("CUFFT calculation completed: %5.3f ms\n", elapsedTime);

    // write filtered image
    uint8_t* output_rgb_image;
    output_rgb_image = (uint8_t*)malloc(width*height);
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            output_rgb_image[i * width + j] = (uint8_t)signal[i * width + j].x;
            if (i < 4 && j < 4) {
                printf("%hhu\n", output_rgb_image[i * width + j]);
            }
        }
    }

    stbi_write_png("image/filtered_dog.png", width, height, 1, output_rgb_image, width);

    // free memory
    hipfftDestroy(plan);
    hipFree(dev_signal);
    hipFree(dev_filter_kernel);
    free(signal);
    free(filter_kernel);
    stbi_image_free(rgb_image);

    return 0;
}