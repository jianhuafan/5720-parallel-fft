// compile command: /usr/local/cuda-10.0/bin/nvcc -arch=compute_35 cuda_fft.cu -lcublas

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>


#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef float2 Complex;

#define DIM 65536

int main(int argc, char **argv) {
    float elapsedTime = 0;
    hipfftHandle plan;
    hipfftComplex *host_data = (hipfftComplex*)malloc(DIM * sizeof(hipfftComplex));
    hipfftComplex *dev_data;
    hipEvent_t start, stop;

    // feed input
    srand(time(NULL));
    for (int i = 0; i < DIM; i++) {
        host_data[i].x = rand() / (float) RAND_MAX;
        host_data[i].y = 0.0;
    }

    // allocate gpu memory
    hipMalloc((void**)&dev_data, sizeof(hipfftComplex) * DIM);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy input to device
    hipMemcpy(dev_data, host_data, DIM * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    printf("test done!\n");
    return 0;
}