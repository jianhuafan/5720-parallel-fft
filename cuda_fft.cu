// compile command: /usr/local/cuda-10.0/bin/nvcc -arch=compute_35 cuda_fft.cu -lcublas -lcufft

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>


#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef float2 Complex;

#define N 4

__global__ void ComplexMul(hipfftComplex *a, hipfftComplex *b) {
    int i = threadIdx.x;
    a[i].x = a[i].x * b[i].x - a[i].y*b[i].y;
    a[i].y = a[i].x * b[i].y + a[i].y*b[i].x;
}

int main(int argc, char **argv) {
    float elapsedTime = 0;
    hipfftHandle plan;
    int mem_size = N*N * sizeof(hipfftComplex);
    hipfftComplex *signal = (hipfftComplex*)malloc(mem_size);
    hipfftComplex *filter_kernel = (hipfftComplex*)malloc(mem_size);
    hipfftComplex *dev_signal;
    hipfftComplex *dev_filter_kernel;
    hipEvent_t start, stop;

    // feed input
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            signal[i * N + j].x = rand() / (float) RAND_MAX;
            signal[i * N + j].y = 0.0;
            filter_kernel[i * N + j].x = rand() / (float) RAND_MAX;
            filter_kernel[i * N + j].y = 0.0;
        }
    }

    // allocate gpu memory
    hipMalloc((void**)&dev_signal, mem_size);
    hipMalloc((void**)&dev_filter_kernel, mem_size);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // copy input to device
    hipMemcpy(dev_signal, signal, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_filter_kernel, filter_kernel, mem_size, hipMemcpyHostToDevice);
    
    // create cufft plan
    hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

    // perform 2dfft
    hipfftExecC2C(plan, dev_signal, dev_signal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, dev_filter_kernel, dev_filter_kernel, HIPFFT_FORWARD);

    // perform multiplication
    ComplexMul <<<32, 256>>>(dev_signal, dev_filter_kernel);

    // perform inverse 2dfft
    hipfftExecC2C(plan, dev_signal, dev_signal, HIPFFT_BACKWARD);

    // get calculation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // show results
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("DATA: %3.1f %3.1f \n", signal[i].x, signal[i].y);
        }
    }
    printf("CUFFT calculation completed: %5.3f ms\n", elapsedTime);

    // free memory
    hipfftDestroy(plan);
    hipFree(dev_signal);
    hipFree(dev_filter_kernel);
    free(signal);
    free(filter_kernel);

    return 0;
}